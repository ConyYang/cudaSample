#include "hip/hip_runtime.h"
// This exercise is for student to get familiarized with passing data between host and device
#include <stdio.h>

__global__ 
void vector_add(int *d_c, int *d_a, int *d_b, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_c[i] = d_a[i] + d_b[i];
    //printf("GPU[%d] done!\n", i);
    }


int main(void){
   int N = 4;
   int a[N] = {22, 13, 16,  5};
   int b[N] = { 5, 22, 17, 37};
   int c[N];

   int *d_a, *d_b, *d_c;

   hipMalloc((void**)&d_a, sizeof(int)*N);
   hipMalloc((void**)&d_b, sizeof(int)*N);
   hipMalloc((void**)&d_c, sizeof(int)*N);

   hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
   hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

   vector_add<<<N,1>>>(d_c, d_a, d_b, N); // N (4) threads

   hipMemcpy(c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost);

   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

   printf("A = [%2d  %2d  %2d  %2d]\n", a[0], a[1], a[2], a[3]);
   printf("B = [%2d  %2d  %2d  %2d]\n", b[0], b[1], b[2], b[3]);
   printf("C = [%2d  %2d  %2d  %2d]\n", c[0], c[1], c[2], c[3]);
   return 0;
   }
