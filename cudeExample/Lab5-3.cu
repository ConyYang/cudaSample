// This exercise is for student to learn about data sharing and synchronization between threads


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 4  // number of elements in vector

__global__ 
void vector_mac(int *d_c, int *d_a, int *d_b, int n){
   __shared__ int tmp[N];      // shared memory

   int i = blockIdx.x * blockDim.x + threadIdx.x;
   tmp[i] = d_a[i] * d_b[i];

   __syncthreads(); // not really necessary for this simple program
   if (i==0){ // use thread 0 to perform the summation
      int sum = 0;
      for (int j = 0; j < n; j++)
         sum = sum + tmp[j];
       *d_c = sum; 
      }
    }

int main(void){
   int a[N] = {22, 13, 16,  5};
   int b[N] = { 5, 22, 17, 37};
   int c[1];

   int *d_a, *d_b, *d_c;
   hipMalloc((void**)&d_a, sizeof(int)*N);
   hipMalloc((void**)&d_b, sizeof(int)*N);
   hipMalloc((void**)&d_c, sizeof(int));

   hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
   hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

   vector_mac<<<1,N>>>(d_c, d_a, d_b, N); // 1 thread block with N (4) threads

   hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);

   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

   printf("A = [%2d  %2d  %2d  %2d]\n", a[0], a[1], a[2], a[3]);
   printf("B = [%2d  %2d  %2d  %2d]\n", b[0], b[1], b[2], b[3]);
   printf("Answer = %d\n", c[0]);
   return 0;
   }
